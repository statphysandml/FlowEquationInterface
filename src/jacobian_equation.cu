#include "hip/hip_runtime.h"
#include <flowequations/jacobian_equation.hpp>


namespace flowequations {
    devdat::DevDatC compute_jacobian_elements(const devdat::DevDatC &coordinates, JacobianEquationsWrapper * jacobian_equations)
    {
        const uint dim = coordinates.dim_size();
        auto number_of_coordinates = coordinates.n_elems();
        devdat::DevDatC jacobian_elements(pow(dim, 2), number_of_coordinates);
        // Evaluate jacobian for each element separately
        for(auto matrix_idx = 0; matrix_idx < pow(dim, 2); matrix_idx ++) {
            (*jacobian_equations)(jacobian_elements[matrix_idx], coordinates, matrix_idx);
        }
        return std::move(jacobian_elements);
    }


    void compute_jacobian_elements(const devdat::DevDatC &coordinates, devdat::DevDatC &jacobian_elements, JacobianEquationsWrapper * jacobian_equations)
    {
        // Evaluate jacobian for each element separately
        for(auto matrix_idx = 0; matrix_idx < pow(coordinates.dim_size(), 2); matrix_idx ++) {
            (*jacobian_equations)(jacobian_elements[matrix_idx], coordinates, matrix_idx);
        }
    }
}
